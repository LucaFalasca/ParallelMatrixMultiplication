#include "hip/hip_runtime.h"
// 
// Author: Luca Falasca
//

// Computes matrix-matrix product.
//

#include <iostream>

#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers
#include <hip/hip_runtime_api.h>
#include <math.h>

#define _DEBUG
// Simple 1-D thread block
// Size should be at least 1 warp 
#define BD 2
#define BD2 24
#define COLS 2

const dim3 BLOCK_DIM(BD);

// Simple CPU implementation of matrix addition.
void CpuMatrixVector(int m, int k, int n, const float* A, const float* x, float* y) {
  float t=0.0;
  for (int row = 0; row < m; ++row) {
    for(int i = 0; i < n; i++){
      t = 0.0;
      for (int col = 0; col < k; ++col) {
        int idx = row * k + col;
        int icx = i * k + col;
        t += A[idx] * x[icx];
        #ifdef _DEBUG
        printf("CPU - A[%d]: %f --- B[%d]: %f\n", idx, A[idx], icx, x[icx]); 
        #endif
      }
      y[i + row * n] += t;
    }
  }
}

// GPU implementation of matrix_vector product using a block of threads for
// each row. 
__device__ void rowReduce(volatile float *sdata, int tid) {
  #ifdef _DEBUG
  if(blockIdx.x == 0){
    printf("sdata[%d]: %f\n", tid, sdata[tid]);
    printf("sdata[%d + 16]: %f\n", tid, sdata[tid + 16]);
    printf("sdata[%d + 8]: %f\n", tid, sdata[tid + 8]);
    printf("sdata[%d + 4]: %f\n", tid, sdata[tid + 4]);
    printf("sdata[%d + 2]: %f\n", tid, sdata[tid + 2]);
    printf("sdata[%d + 1]: %f\n", tid, sdata[tid + 1]);
  }
  #endif
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid +  8];
  sdata[tid] += sdata[tid +  4];
  sdata[tid] += sdata[tid +  2];
  sdata[tid] += sdata[tid +  1];
}

__device__ void rowReduce2(volatile float *sdata, int tid, int s) {
  #ifdef _DEBUG
  
  if(blockIdx.x == 0){
    printf("row->%d sdata[%d]: %f\n", blockIdx.x, tid, sdata[tid]);
    printf("row->%d sdata[%d + 16]: %f\n", blockIdx.x, tid, sdata[tid + 16]);
    printf("row->%d sdata[%d + 8]: %f\n", blockIdx.x, tid, sdata[tid + 8]);
    printf("row->%d sdata[%d + 4]: %f\n", blockIdx.x, tid, sdata[tid + 4]);
    printf("row->%d sdata[%d + 2]: %f\n", blockIdx.x, tid, sdata[tid + 2]);
    printf("row->%d sdata[%d + 1]: %f\n", blockIdx.x, tid, sdata[tid + 1]);
  }
  
  #endif
  switch(s){
  case 16:  sdata[tid] += sdata[tid + 16];
  case  8:  sdata[tid] += sdata[tid +  8];
  case  4:  sdata[tid] += sdata[tid +  4];
  case  2:  sdata[tid] += sdata[tid +  2];
  case  1:  sdata[tid] += sdata[tid +  1];
  }
}

__global__ void gpuMatrixMatrix(int m, int k, int n, const float* A,
				const float* B, float* C) {
  __shared__ float aux[COLS][BD];
  int tc     = threadIdx.x;
  int row    = blockIdx.x;
  int s = min(16,BD/2);
  int n_cols = COLS;
  __shared__ float a_shared[BD];
  
  if (row < m) {
    // Itero per ogni blocco di colonne della matrice B
    for(int i = 0; i < n; i+= COLS){
      // Se l'ultimo blocco di colonne è minore di COLS aggiorno il numero di colonne del blocco
      if(i + COLS > n) n_cols = n % COLS; 

      // Inizializzo la matrice ausiliaria per la reduce
      for(int j = 0; j < n_cols; j++) aux[j][tc] = 0.0;


      /*
        Ogni processo si calcola il prodotto tra il valore della riga di A e gli elementi corrispondenti 
        delle colonne del blocco di colonne di B 
      */
      int idxm = row*k+tc;  //indice dell'elemento della riga A associato al processo

      // Itero sui blocchi della riga di A 
      for (int ic= tc;  ic<k; ic += blockDim.x) {
        //Ogni processo ha in memoria condivisa il valore del'elemento della riga di A corrispondente
        a_shared[tc] = A[idxm];
        // Itero sulle colonne del blocco di colonne di B
        for(int j = 0; j < n_cols; j++){
          // Calcolo l'indice dell'elemento di B corrispondente
          int icx = (i + j) * k + ic;
          aux[j][tc] += a_shared[tc]*B[icx];
          #ifdef _DEBUG
          printf("{Blocco %d} P%d-A[%d]: %f --- B[%d]: %f\n", row, tc, idxm, A[idxm], icx, B[icx]); 
          #endif
        }
        idxm +=  blockDim.x;
      }
      __syncthreads();

      //print aux matrix
      #ifdef _DEBUG
      if(tc == 0){
        for(int j = 0; j < n_cols; j++){
          for(int k = 0; k < BD; k++){
            printf("[BEFORE]row->%d aux[%d][%d]: %f\n", row, j, k, aux[j][k]);
          }
        }
      }
      #endif

      // Reduce
      for(int j = 0; j < n_cols; j++){
        for (int s2=BD/2; s2 >=32; s2 >>=1){
          if (tc<s2)
            aux[j][tc] += aux[j][tc+s2]; 
          __syncthreads();
        }
      }

      for(int j = 0; j < n_cols; j++){
        if (tc < s) rowReduce2(&(aux[j][0]),tc,s);

        if (tc == 0){
          #ifdef _DEBUG
          if(1)
          printf("[AFTER REDUCE]row->%d aux[%d][%d]: %f\n", row, j, tc, aux[j][tc]);
          printf("[FINAL] y[%d] = aux[%d][%d]: %f\n", i + j + row * n, j, tc, aux[j][tc]);
          #endif

          C[i + j + row * n] += aux[j][tc];
        }
      }
    
    }
  }
}

__global__ void gpuMatrixVectorV4(int m, int k, int n, const float* A,
				const float* B, float* y) {
  __shared__ float aux[BD2][BD];
  int tc     = threadIdx.x;
  int row    = blockIdx.x;
  int s = min(16,BD/2);
  int size = BD2;
  __shared__ float a_shared[BD];
  
  if (row < m) {
    
    // Itero per ogni colonna della matrice B (n)
    for(int i = 0; i < n; i+= BD2){
      if(i + BD2 > n) size = n % BD2;
      for(int j = 0; j < size; j++) aux[j][tc] = 0.0;
      int idxm = row*k+tc;
      for (int ic= tc;  ic<k; ic += blockDim.x) {
        a_shared[tc] = A[idxm];
        for(int j = 0; j < size; j++){
          int icx = i * k + ic + j * k;
          aux[j][tc] += a_shared[tc]*B[icx];
          #ifdef _DEBUG
          printf("{Blocco %d} P%d-A[%d]: %f --- B[%d]: %f\n", row, tc, idxm, A[idxm], icx, B[icx]); 
          #endif
        }
        idxm +=  blockDim.x;
      }
      
      
    
      __syncthreads();
      //print aux matrix
      #ifdef _DEBUG
      if(tc == 0){
        for(int j = 0; j < size; j++){
          for(int k = 0; k < BD; k++){
            printf("[BEFORE]row->%d aux[%d][%d]: %f\n", row, j, k, aux[j][k]);
          }
        }
      }
      #endif
      for(int j = 0; j < size; j++){
        for (int s2=BD/2; s2 >=32; s2 >>=1){
          if (tc<s2)
            aux[j][tc] += aux[j][tc+s2]; 
          __syncthreads();
        }
      }

      for(int j = 0; j < size; j++){
        if (tc < s) rowReduce2(&(aux[j][0]),tc,s);

        if (tc == 0){
          #ifdef _DEBUG
          if(1)
          printf("[AFTER REDUCE]row->%d aux[%d][%d]: %f\n", row, j, tc, aux[j][tc]);
          printf("[FINAL] y[%d] = aux[%d][%d]: %f\n", i + j + row * n, j, tc, aux[j][tc]);
          #endif

          y[i + j + row * n] += aux[j][tc];
        }
      }
    
    }
  }
}

__global__ void gpuMatrixVectorV3(int m, int k, int n, const float* A,
				const float* B, float* y) {
  __shared__ float aux[BD][BD2];
  int tc     = threadIdx.x;
  int row    = blockIdx.x;
  int s = min(16,BD/2);
  int size = BD2;
  
  if (row < m) {
    
    // Itero per ogni colonna della matrice B (n)
    for(int i = 0; i < n; i+= BD2){
      if(i + BD2 > n) size = n % BD2;
      for(int j = 0; j < size; j++){
        int idxm = row*k+tc;
        float t  = 0.0;
        int q = 0;
        aux[j][tc] = 0.0;
        for (int ic= tc;  ic<k; ic += blockDim.x) {
          int icx = i * k + ic + j * k;
          t += A[idxm]*B[icx];
          #ifdef _DEBUG
          printf("{Blocco %d} P%d-A[%d]: %f --- B[%d]: %f\n", row, tc, idxm, A[idxm], icx, B[icx]); 
          #endif
          q++;
          idxm +=  blockDim.x;
        }
        aux[j][tc] = t;
      }
      
      
    
      __syncthreads();
      //print aux matrix
      #ifdef _DEBUG
      if(tc == 0){
        for(int j = 0; j < size; j++){
          for(int k = 0; k < BD; k++){
            printf("[BEFORE]row->%d aux[%d][%d]: %f\n", row, j, k, aux[j][k]);
          }
        }
      }
      #endif
      for(int j = 0; j < size; j++){
        for (int s2=BD/2; s2 >=32; s2 >>=1){
          if (tc<s2)
            aux[j][tc] += aux[j][tc+s2]; 
          __syncthreads();
        }
      }

      for(int j = 0; j < size; j++){
        if (tc < s) rowReduce2(&(aux[j][0]),tc,s);

        if (tc == 0){
          #ifdef _DEBUG
          if(1)
          printf("[AFTER REDUCE]row->%d aux[%d][%d]: %f\n", row, j, tc, aux[j][tc]);
          printf("[FINAL] y[%d] = aux[%d][%d]: %f\n", i + j + row * n, j, tc, aux[j][tc]);
          #endif

          y[i + j + row * n] = aux[j][tc];
        }
      }
    
    }
  }
}


__global__ void gpuMatrixVectorV2(int m, int k, int n, const float* A,
				const float* B, float* y) {
  __shared__ float aux[BD];
  extern __shared__ float row_shared[];

  int tc     = threadIdx.x;
  int row    = blockIdx.x;
  int size_shared_vec = abs(k / BD);
  
  if (row < m) {
    
    // Itero per ogni colonna della matrice B (n)
    for(int i = 0; i < n; i++){
      int idxm = row*k+tc;
      float t  = 0.0;
      int q = 0;
      aux[tc] = 0.0;
      int irs = 0;
      //ogni processo prende e moltiplica solo alcuni pezzi della matrice A per i corrispendenti della colonna i di B
      for (int ic= tc;  ic<k; ic += blockDim.x) {
        int icx = i * n + ic;
        irs = q + size_shared_vec * tc + k % BD * (tc != 0);
        if(i == 0){
          //Se sto facendo la riga per la prima colonna allora prendo i valori dalla memoria globale e li metto in shared
          //dato che mi serviranno per le altre colonne
          int row_temp = A[idxm];
          row_shared[irs] = row_temp;
          t+= row_temp*B[icx];
        }
        else{
          //Se sto facendo la riga per le altre colonne allora prendo i valori dalla memoria shared
          t += row_shared[q + size_shared_vec * tc]*B[icx];
          //t += A[idxm]*B[icx];
          #ifdef _DEBUG
          printf("{Blocco %d} P%d-A[%d]: %f --- x[%d]: %f\n", row, tc, idxm, A[idxm], icx, B[icx]); 
          #endif
        }
        q++;
        idxm +=  blockDim.x;
      }
      aux[tc] = t;
    
    
      __syncthreads();
      for (int s=BD/2; s >=32; s >>=1){
        if (tc<s)
          aux[tc] += aux[tc+s]; 
        __syncthreads();
      }
    
      
      if (tc<16) rowReduce(aux,tc);
      
      if (tc == 0)
        y[i + row * n] = aux[tc];
    }
  }
}

__global__ void gpuMatrixVectorV1(int m, int k, int n, const float* A,
				const float* B, float* y) {
  __shared__ float aux[BD];
  int tc     = threadIdx.x;
  int row    = blockIdx.x;
  if (row < m) {
    
    // Starting address of indexing within matrix A
    for(int i = 0; i < n; i++){
      int idxm = row*k+tc;
      float t  = 0.0;
      int q = 0;
      aux[tc] = 0.0;
      for (int ic= tc;  ic<k; ic += blockDim.x) {
        int icx = i * n + ic;
        t += A[idxm]*B[icx];
        #ifdef _DEBUG
          printf("{Blocco %d} P%d-A[%d]: %f --- x[%d]: %f\n", row, tc, idxm, A[idxm], icx, B[icx]); 
          #endif
        q++;
        idxm +=  blockDim.x;
      }
      aux[tc] = t;
    
    
      __syncthreads();
      for (int s=BD/2; s >=32; s >>=1){
        if (tc<s)
          aux[tc] += aux[tc+s]; 
        __syncthreads();
      }
    
      
      if (tc<16) rowReduce(aux,tc);
      
      if (tc == 0)
        y[i + row * n] = aux[tc];
    }
  }
}

int main(int argc, char** argv) {

  if (argc < 3) {
    fprintf(stderr,"Usage: %s  rows cols\n",argv[0]);
  }
  int m=atoi(argv[1]);
  int k=atoi(argv[2]);
  int n=atoi(argv[3]);
  
  
  // ----------------------- Host memory initialisation ----------------------- //

  float* h_A = new float[m * k];
  float* h_B = new float[k * n];
  float* h_y = new float[m * n];
  float* h_y_d = new float[m * n];

  srand(123456);
  #ifdef _DEBUG 
  std::cout << "Matrix A: " << std::endl;
  #endif
  for (int row = 0; row < m; ++row) {
    for (int col = 0; col < k; ++col) {
      int idx = row * k + col;
      h_A[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
      #ifdef _DEBUG 
      std::cout << "|" << h_A[idx] << "";
      #endif
    }
    #ifdef _DEBUG 
    std::cout << "|" << std::endl;
    #endif
    h_y[row] = 0.0;
  }
  #ifdef _DEBUG 
  std::cout << "\nMatrix B:" << std::endl;
  #endif
  for (int col = 0; col < k; ++col) {
    for(int row = 0; row < n; ++row){
      int idx = row * k + col;
      h_B[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
      #ifdef _DEBUG 
      std::cout << "|" << h_B[idx] << "";
      #endif
    }
    #ifdef _DEBUG 
    std::cout << "|"<< std::endl;
    #endif
  }

  memset(h_y, 0, m * n * sizeof(float));
  

  std::cout << "Matrix-vector product: 1D thread block version " << std::endl;
  std::cout << "Test case: [" << m  << "x" << k << "] x ["<< k << "x" << n << "]" << std::endl;
  std::cout << "m = " << m  << " | k = " << k << "| n = "<< n << std::endl;
// ---------------------- Device memory initialisation ---------------------- //

  float *d_A, *d_B, *d_y;

  checkCudaErrors(hipMalloc((void**) &d_A, m * k * sizeof(float)));
  checkCudaErrors(hipMalloc((void**) &d_B, k * n * sizeof(float)));
  checkCudaErrors(hipMalloc((void**) &d_y, m * n * sizeof(float)));

  // Copy matrices from the host (CPU) to the device (GPU).
  checkCudaErrors(hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, m * n * sizeof(float), hipMemcpyHostToDevice));

  // ------------------------ Calculations on the CPU ------------------------- //
  
  float flopcnt=2.e-6*m*k*n;
  
  // Create the CUDA SDK timer.
  
  StopWatchInterface* timer = 0;
  sdkCreateTimer(&timer);
  
  
  timer->start();
  CpuMatrixVector(m, k, n, h_A, h_B, h_y);

  timer->stop();
  float cpuflops=flopcnt/ timer->getTime();
  std::cout << "  CPU time: " << timer->getTime() << " ms." << " GFLOPS " << cpuflops << std::endl;
  

// ------------------------ Calculations on the GPU ------------------------- //

  // Calculate the dimension of the grid of blocks (1D) needed to cover all
  // entries in the matrix and output vector
  const dim3 GRID_DIM(m,1);
  size_t smemSize = k * sizeof(float);
  float gpuflops;

  //printf("size of shared memory: %d\n", smemSize);
  
  timer->reset();
  timer->start();
  gpuMatrixVectorV1<<<GRID_DIM, BLOCK_DIM>>>(m, k, n, d_A, d_B, d_y);
  checkCudaErrors(hipDeviceSynchronize());

  timer->stop();
  gpuflops=flopcnt/ timer->getTime();
  std::cout << "  GPU time global memory: " << timer->getTime() << " ms." << " GFLOPS " << gpuflops<<std::endl;

  float* zero = new float[m * n];
  memset(zero, 0, m * n * sizeof(float));
  checkCudaErrors(hipMemcpy(d_y, zero, m * n * sizeof(float), hipMemcpyHostToDevice));
  
  printf("size of shared memory: %d\n", BD * BD2 * sizeof(float) + BD * sizeof(float));
  timer->reset();
  timer->start();
  hipProfilerStart();
  gpuMatrixVectorV4<<<GRID_DIM, BLOCK_DIM>>>(m, k, n, d_A, d_B, d_y);
  hipProfilerStop();
  checkCudaErrors(hipDeviceSynchronize());

  timer->stop();
  gpuflops=flopcnt/ timer->getTime();
  std::cout << "  GPU time shared memory: " << timer->getTime() << " ms." << " GFLOPS " << gpuflops<<std::endl;

  checkCudaErrors(hipMemcpy(d_y, zero, m * n * sizeof(float), hipMemcpyHostToDevice));

  printf("size of shared memory: %d\n", BD * BD2 * sizeof(float) + BD * sizeof(float));
  timer->reset();
  timer->start();
  hipProfilerStart();
  gpuMatrixMatrix<<<GRID_DIM, BLOCK_DIM>>>(m, k, n, d_A, d_B, d_y);
  hipProfilerStop();
  checkCudaErrors(hipDeviceSynchronize());

  timer->stop();
  gpuflops=flopcnt/ timer->getTime();
  std::cout << "  GPU time shared memory: " << timer->getTime() << " ms." << " GFLOPS " << gpuflops<<std::endl;

  

  // Download the resulting vector d_y from the device and store it in h_y_d.
  checkCudaErrors(hipMemcpy(h_y_d, d_y, m*n*sizeof(float),hipMemcpyDeviceToHost));

  //print h_y_d
  #ifdef _DEBUG
  std::cout << "Matrix-vector product: 1D thread block version " << std::endl;
  std::cout << "Test case: " << m  << " x " << n << std::endl;
  std::cout << "Matrix y_d: " << std::endl;
  for (int row = 0; row < m; ++row) {
    for (int col = 0; col < n; ++col) {
      int idx = row * n + col;
      std::cout << "|" << h_y_d[idx] << "";
    }
    std::cout << "|" << std::endl;
  }
  #endif

  #ifdef _DEBUG
  std::cout << "Matrix-vector product: CPU version " << std::endl;
  std::cout << "Test case: " << m  << " x " << n << std::endl;
  std::cout << "Matrix y_d: " << std::endl;
  for (int row = 0; row < m; ++row) {
    for (int col = 0; col < n; ++col) {
      int idx = row * n + col;
      std::cout << "|" << h_y[idx] << "";
    }
    std::cout << "|" << std::endl;
  }
  #endif

  // Now let's check if the results are the same.
  float reldiff = 0.0f;
  float diff = 0.0f;
  
  for (int row = 0; row < m; ++row) {
    for (int col = 0; col < n; ++col) {
      float maxabs = std::max(std::abs(h_y[row + col * n]),std::abs(h_y_d[row + col * n]));
      if (maxabs == 0.0) maxabs=1.0;
      reldiff = std::max(reldiff, std::abs(h_y[row + col * n] - h_y_d[row + col * n])/maxabs);
      diff = std::max(diff, std::abs(h_y[row + col * n] - h_y_d[row + col * n]));
      //std::cout << row<<" "<<h_y[row]<<" "<<h_y_d[row] <<std::endl;
    }
  }
  std::cout << "Max diff = " << diff << "  Max rel diff = " << reldiff << std::endl;
  // Rel diff should be as close as possible to unit roundoff; float
  // corresponds to IEEE single precision, so unit roundoff is
  // 1.19e-07
  // 

// ------------------------------- Cleaning up ------------------------------ //

  delete timer;

  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_y));
  
  delete[] h_A;
  delete[] h_B;
  delete[] h_y;
  delete[] h_y_d;
  
  return 0;
}
